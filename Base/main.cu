#include "hip/hip_runtime.h"
/* Copyright (c) 2005 Agorics; see MIT_License in this directory
or http://www.opensource.org/licenses/mit-license.html */

/* This program is used to create an ELF object file containing
 * primordial nodes, primordial pages and the control structures for
 * the pages. The pages are either zero filled pre-allocated storage,
 * domain program text and data or flat file data.
 * The resulting .o file will contain three sections and 6 symbols.
 *
 *	Sections:
 *		.prim_nodes
 *		.prim_plist
 *		.prim_pages
 *
 *	Symbols:
 *		prim_nodecnt
 *		prim_node
 *		prim_plistcnt
 *		prim_plist
 *		prim_pagecnt
 *		prim_pages
 */

#include <stdio.h>
#include <stdarg.h>
#include <string.h>
#include <fcntl.h>
#include <libelf.h>
#include <unistd.h>
#include "kktypes.h"
#include "cvt.h"
#include "itemdefh.h"
#include "item.h"
#include "disknodh.h"

/* Macros */

#define PAGESIZE 4096


/* Structure to record info for a given elf section */

typedef struct scn_info {
	Elf_Scn *scn;		/* section descriptor */
	Elf_Data *data;		/* section data */
	Elf32_Shdr *shdr;	/* section header */
	uint_t ndx; 		/* section index */
	uint_t offset;		/* used by string & symbol sections */
	struct scn_info *link;	/* used by symbol section */
} scninfo_t;


/* Structure to record info for a given elf object file */

typedef struct {
	Elf *elf;			/* elf descriptor */
	Elf32_Ehdr *ehdr;		/* elf header */
	scninfo_t *shstrscn_info;	/* section header str table */
	scninfo_t *symscn_info;		/* symbol table */
	scninfo_t *cntscn_info;		/* section to hold counts */
} elf_info_t;

/* enumeration to list the different file types we are interested in */

typedef enum {
	FT_NOFILE,
	FT_PLAIN,
	FT_EXEC
} filetype_t;


/* Static support functions */

static int readfile(char *, char *, int);
static filetype_t filetype(char *filename);
static void elfload(char *, char *, char *, ulong_t);
static void elfunload(const char *, int, int, int, int, int, int);
static elf_info_t *elf_outfile_init(char *filename);
static void elf_outfile_fini(elf_info_t *einfop);
static scninfo_t *make_strtab(Elf *elf);
static scninfo_t *make_symtab(Elf *elf);
static scninfo_t *make_cntscn(Elf *elf);
static void make_primordial_node_scn(elf_info_t *einfop, char *buf,
	ulong_t count);
static void make_primordial_plist_scn(elf_info_t *einfop, char *buf,
	ulong_t count);
static void make_primordial_page_scn(elf_info_t *einfop, char *buf,
	size_t size);
static void error(char *fmt, ...);
static void set_search_path(char *pathstr);


/* External support functions defined */
int file_countpages(char *filename, int filetype);
char *getfullname(char *);

extern prim_info_t *def_initial_items();

/* External support functions used */

extern prim_info_t *def_initial_items();


/* Static support data */

static char *cmdname;


/* External support data */

extern plist_t plist[];
extern DiskNode_t nodes[];
int verbose;


/* primbuilder [-P search_list] [-o output_file] [-v]
 * search_list is a colon separated list of directories in which to
 * search for domains.
 * output_file is the file in which to write the primordial nodes and
 * pages.
 * -v can be used to print a report of the types of nodes and pages
 * generated.
 */
int
main(int argc, char *argv[])
{
	ulong_t i, total_page_size = 0;
	char *pagebase, *pagep;
	prim_info_t *prim_infop;
	elf_info_t *einfop;
	char *outfile_name = "primordial.o";

	cmdname = argv[0];

	/* Check for domain search path and output file name */
	for (i = 1; i < argc; i++) {
		if (strncmp(argv[i], "-P", 2) == 0) {
			/* have a path */
			if (argv[i][2] == '\0')
				set_search_path(argv[i+1]);
			else
				set_search_path(argv[i]+2);
		} else if (strncmp(argv[i], "-o", 2) == 0) {
			if (argv[i][2] == '\0')
				outfile_name = argv[i+1];
			else
				outfile_name = argv[i]+2;
		} else if (strncmp(argv[i], "-v", 2) == 0) 
			verbose = 1;
	}

	/* Populate the plist and node arrays */
	prim_infop = def_initial_items();

	/* Determine how much space we need, grab it and zero it out. */
	for (i = 0; i < prim_infop->plistcnt; i++)
		total_page_size += plist[i].number * PAGESIZE;

	if ((pagebase = (char *)malloc(total_page_size)) == NULL)
		error("main: Memory allocation failure");

	memset(pagebase, 0, total_page_size);	/* zero the pages */

	/* Now loop through and fill the memory allocated */
	pagep = pagebase;
	for (i = 0; i < prim_infop->plistcnt; i++) {
		char *fname = plist[i].filename;
		size_t size = plist[i].number * PAGESIZE;

		switch (plist[i].first) {
		case 0: 
			/* load ELF executable */
			if (fname != NULL && *fname != '\0')
				elfload(fname, (char *)pagep, NULL, 
					size);
			break;

		default:  {
			/* read entire file */
			ulong_t length;

			if (fname != NULL && *fname != '\0') {
				length = readfile(fname, (char *)pagep, 
					size);
				if (plist[i].lengthplace != 0)
					long2b(length, plist[i].lengthplace, 6);
			}
			break;
		}
		} /* end switch */
		pagep += size;
	}

	/* initialize the outfile */
	einfop = elf_outfile_init(outfile_name);

	/* Write the .prim_nodes section  - don't bother with the
	 * first node since it is zero'd out.
	 */
	make_primordial_node_scn(einfop, (char *)(nodes + 1), 
		prim_infop->nodecnt);

	/* Write the .prim_plist section */
	make_primordial_plist_scn(einfop, (char *)plist, 
		prim_infop->plistcnt);

	/* Write the .prim_pages section */
	make_primordial_page_scn(einfop, pagebase, total_page_size);

	elf_outfile_fini(einfop);
	exit(0);
}

/* filetype() attempts to determine the type of a file. It will return
 * FT_NOFILE is the filename is NULL or empty. It will return FT_EXEC
 * if the file is an elf executable. Otherwise it returns FT_PLAIN.
 */
filetype_t
filetype(char *filename)
{
	int fildes;
	Elf *elf;
	Elf32_Ehdr *ehdr;

	if (filename == NULL || *filename == '\0')
		return FT_NOFILE;
	
	if (elf_version(EV_CURRENT) == EV_NONE) 
		error("filetype: Invalid elf version");

	if ((fildes = open(filename, O_RDONLY)) == -1)
		error("filetype: Cannot open file %s", filename);

	elf = elf_begin(fildes, ELF_C_READ, (Elf *)NULL);

	if ((ehdr = elf32_getehdr(elf)) == NULL)
		return FT_PLAIN;

	if (ehdr->e_type == ET_EXEC)
		return FT_EXEC;
	else
		return FT_PLAIN;
}

/*
 * Note: Currently symbuf is not used. At some point we may find it
 * useful to load in the symbol table and string table.
 */

static void
elfload(char *filename, char *codebuf, char *symbuf, ulong_t maxsize)
{
	int fildes;
	Elf *elf;
	Elf32_Phdr *php, *phtable, *phtable_end;
	Elf32_Ehdr *ehdr;
	char *rawptr;
	size_t total_bytes_copied = 0;
	int found_loadable = 0;

	if (elf_version(EV_CURRENT) == EV_NONE) 
		error("elfload: Invalid elf version");

	if ((fildes = open(filename, O_RDONLY)) == -1)
		error("elfload: Cannot open file %s", filename);

	elf = elf_begin(fildes, ELF_C_READ, (Elf *)NULL);

	if ((ehdr = elf32_getehdr(elf)) == NULL)
		error("elfload: Cannot read elf header: %s", filename);

	if (ehdr->e_type != ET_EXEC)
		error("elfload: Not an elf executable: %s", filename);

	if ((phtable = elf32_getphdr(elf)) == NULL)
		error("elfload: missing program header table: %s",
			filename);

	phtable_end = phtable + ehdr->e_phnum;
	php = phtable;
	rawptr = elf_rawfile(elf, 0);

	/* Loops through the segments. We should only find one loadable
	 * segment. If we find more than one, panic since we are
	 * assuming than anything we are ELF loading can be directly
	 * mapped by the kernel. If there are multiple loadable
	 * segments, the mapping will probably fail.
	 */
	while (php < phtable_end) {
		if (php->p_type == PT_LOAD) {
			if (found_loadable++)
				error("elfload: multiple loadable segments in mapped file\n");
			if ((total_bytes_copied + php->p_memsz) > maxsize)
				error("elfload: file larger than "
					"allocated size: %s", filename);

			/* Copy file data */
			memcpy(codebuf, rawptr + php->p_offset, 
				php->p_filesz);
			if (php->p_memsz > php->p_filesz)
				memset(codebuf + php->p_filesz, 0, 
					php->p_memsz - php->p_filesz);
			total_bytes_copied += php->p_memsz;
			codebuf += total_bytes_copied;
		}
		php++;
	}
	/* Zero out remainder of buffer */
	if (total_bytes_copied < maxsize)
		memset(codebuf, 0, maxsize - total_bytes_copied);
	return;
}

/* file_countpages() is used to determine how many pages will be needed
 * to hold the loadable segments from an executable.
 */
int
file_countpages(char *filename, int filetype)
{
	int fildes;
	Elf *elf;
	Elf32_Phdr *php, *phtable, *phtable_end;
	Elf32_Ehdr *ehdr;
	char *rawptr;
	size_t total_bytes = 0;
	int found_loadable = 0;

	if (elf_version(EV_CURRENT) == EV_NONE) 
		error("file_countpages: Invalid elf version");

	if ((fildes = open(filename, O_RDONLY)) == -1)
		error("file_countpages: Cannot open file %s", filename);

	elf = elf_begin(fildes, ELF_C_READ, (Elf *)NULL);

	if (filetype != 0) { /* just return the size of the file */
		int size;

		if ((size = lseek(fildes, 0, SEEK_END)) == -1)
			error("file_countpages: Cannot seek in file: %s\n",
				filename);
		return ((size + (PAGESIZE-1)) / PAGESIZE);
	}

	/* Should be an ELF executable, make sure */
	if ((ehdr = elf32_getehdr(elf)) == NULL ||
		(ehdr->e_type != ET_EXEC) ||
		(phtable = elf32_getphdr(elf)) == NULL) {

		error("file_countpages: expecting ELF executable, didn't get one\n");
	}
	
	/* It's an ELF executable - make sure there's only one
	 * loadable segment (we're using this as a test that it's
	 * mappable).
	 */
	phtable_end = phtable + ehdr->e_phnum;
	php = phtable;
	rawptr = elf_rawfile(elf, 0);

	/* Loops through the segments, counting sizes of LOADable 
	 * segments.
	 */
	while (php < phtable_end) {
		if (php->p_type == PT_LOAD) {
			if (found_loadable++)
				error("file_countpages: multiple loadable segments in mapped file\n");
			total_bytes = php->p_memsz;
		}
		php++;
	}
	return (int)((total_bytes + (PAGESIZE-1)) / PAGESIZE);
}

static uint_t add_str(scninfo_t *si, char *string);
static void add_symbol(scninfo_t *sym_si, char *string, int value, 
	size_t size, Elf_Scn *targ_scn);
static void set_tab_size(scninfo_t *);

static elf_info_t *
elf_outfile_init(char *filename)
{
	elf_info_t *einfop;
	int fd;
	scninfo_t *symstrscn_info;

	if (elf_version(EV_CURRENT) == EV_NONE) 
		error("elf_outfile_init: Invalid elf version");

	if (filename == NULL) 
		error("elf_outfile_init: NULL output file name");

	if ((fd = open(filename, O_RDWR|O_TRUNC|O_CREAT, 0666)) == -1) 
		error("elf_outfile_init: Cannot open file: %s", 
			filename);

	if ((einfop = (elf_info_t *)malloc(sizeof(elf_info_t))) == NULL)
		error("elf_outfile_init: Memory allocation failure");

	if ((einfop->elf = elf_begin(fd, ELF_C_WRITE, (Elf *)0)) == 0) 
		error("elf_outfile_init: Cannot build output file: %s",
			filename);

	einfop->ehdr = elf32_newehdr(einfop->elf);

	/* Mark this as a SPARC object */
	einfop->ehdr->e_machine = EM_SPARC;

	/* Mark this as a relocatable object */
	einfop->ehdr->e_type = ET_REL;

	/* create section header string table section */
	einfop->shstrscn_info = make_strtab(einfop->elf);
	einfop->shstrscn_info->shdr->sh_name = 
		add_str(einfop->shstrscn_info, ".shstrtab");
	einfop->ehdr->e_shstrndx = einfop->shstrscn_info->ndx;

	/* create symbol table string table section */
	symstrscn_info = make_strtab(einfop->elf);
	symstrscn_info->shdr->sh_name = 
		add_str(einfop->shstrscn_info, ".strtab");

	/* create symbol table section */
	einfop->symscn_info = make_symtab(einfop->elf);
	einfop->symscn_info->shdr->sh_name = 
		add_str(einfop->shstrscn_info, ".symtab");
	einfop->symscn_info->link = symstrscn_info;
	einfop->symscn_info->shdr->sh_link = symstrscn_info->ndx;

	/* create .count section - for holding counts */
	einfop->cntscn_info = make_cntscn(einfop->elf);
	einfop->cntscn_info->shdr->sh_name = 
		add_str(einfop->shstrscn_info, ".count");

	return einfop;
}

static void
elf_outfile_fini(elf_info_t *einfop)
{
	/* Update the size of the section header string table and
	 * finish writing out the file.
	 */

	/* set size of section header string table */
	set_tab_size(einfop->shstrscn_info);

	/* set size of symbol table */
	set_tab_size(einfop->symscn_info);

	/* set size of symbol table's string table */
	set_tab_size(einfop->symscn_info->link);

	elf_update(einfop->elf, ELF_C_WRITE);
	elf_end(einfop->elf);
}

#define STRTAB_SIZE 1000

static scninfo_t *
make_strtab(Elf *elf)
{
	scninfo_t *si;

	if ((si = (scninfo_t *)malloc(sizeof(scninfo_t))) == NULL)
		error("make_strtab: Memory allocation failure");

	si->scn = elf_newscn(elf);
	si->shdr = elf32_getshdr(si->scn);
	si->ndx = elf_ndxscn(si->scn);
	si->data = elf_newdata(si->scn);
	if ((si->data->d_buf = (char *)malloc(STRTAB_SIZE)) == NULL)
		error("make_strtab: Memory allocation failure");

	si->data->d_size = STRTAB_SIZE;
	si->data->d_off = 0;
	si->data->d_align = 1;
	si->shdr->sh_type = SHT_STRTAB;
	si->offset = 1; /* first byte should be 0 */
	return si;
}

#define SYMTAB_SIZE 1000

static scninfo_t *
make_symtab(Elf *elf)
{
	scninfo_t *si;

	if ((si = (scninfo_t *)malloc(sizeof(scninfo_t))) == NULL)
		error("make_symtab: Memory allocation failure");

	si->scn = elf_newscn(elf);
	si->shdr = elf32_getshdr(si->scn);
	si->ndx = elf_ndxscn(si->scn);
	si->data = elf_newdata(si->scn);
	if ((si->data->d_buf = (char *) malloc(SYMTAB_SIZE)) == NULL)
		error("make_symtab: Memory allocation failure");

	si->data->d_size = SYMTAB_SIZE;
	si->data->d_off = 0;
	si->data->d_align = 4;
	si->shdr->sh_type = SHT_SYMTAB;
	si->shdr->sh_info = 1; /* point past zero sym */
	si->offset = sizeof(Elf32_Sym); /* first symbol is NULL */
	return si;
}

static scninfo_t *
make_cntscn(Elf *elf)
{
	scninfo_t *si;

	if ((si = (scninfo_t *)malloc(sizeof(scninfo_t))) == NULL)
		error("make_cntscn: Memory allocation failure");

	si->scn = elf_newscn(elf);
	si->shdr = elf32_getshdr(si->scn);
	si->ndx = elf_ndxscn(si->scn);
	si->shdr->sh_flags = SHF_ALLOC + SHF_WRITE;
	si->shdr->sh_type = SHT_PROGBITS;
	return si;
}

static void add_cnt(scninfo_t *si, char *cntname, size_t size, 
	uint_t value, scninfo_t *sym_si);

static void
make_primordial_node_scn(elf_info_t *einfop, char *buf, ulong_t count)
{
	Elf_Scn *scn;		/* section descriptor */
	Elf_Data *data;		/* section data */
	Elf32_Shdr *shdr;	/* section header */
	Elf *elf = einfop->elf;
	size_t size = count * sizeof(DiskNode_t);

	scn = elf_newscn(elf);
	shdr = elf32_getshdr(scn);

	/* create object in the .count scn to hold the node count */
	add_cnt(einfop->cntscn_info, "prim_nodecnt", sizeof(int *), 
		count, einfop->symscn_info);

	/* create a symbol to point to the node data */
	add_symbol(einfop->symscn_info, "prim_nodes", 0, size, scn);

	/* create a data buffer to hold the nodes */
	data = elf_newdata(scn);
	if ((data->d_buf = (char *) malloc(size)) == NULL)
		error("make_primordial_node_scn: "
			"Memory allocation failure");

	memcpy(data->d_buf, buf, size);
	data->d_size = size;
	data->d_off = 0;
	data->d_align = 0x1000;
	shdr->sh_type = SHT_PROGBITS;
	shdr->sh_flags = SHF_ALLOC + SHF_WRITE;
	shdr->sh_name = 
		add_str(einfop->shstrscn_info, ".prim_nodes");
	return;
}

static void
make_primordial_plist_scn(elf_info_t *einfop, char *buf, ulong_t count)
{
	Elf_Scn *scn;		/* section descriptor */
	Elf_Data *data;		/* section data */
	Elf32_Shdr *shdr;	/* section header */
	Elf *elf = einfop->elf;
	size_t size = count * sizeof(plist_t);

	scn = elf_newscn(elf);
	shdr = elf32_getshdr(scn);

	/* create object in the .count scn to hold the plist count */
	add_cnt(einfop->cntscn_info, "prim_plistcnt", sizeof(int *), 
		count, einfop->symscn_info);

	/* create a symbol to point to the plist data */
	add_symbol(einfop->symscn_info, "prim_plist", 0, size, scn);

	data = elf_newdata(scn);
	if ((data->d_buf = (char *) malloc(size)) == NULL)
		error("make_primordial_plist_scn: "
			"Memory allocation failure");

	memcpy(data->d_buf, buf, size);
	data->d_size = size;
	data->d_off = 0;
	data->d_align = 0x1000;
	shdr->sh_type = SHT_PROGBITS;
	shdr->sh_flags = SHF_ALLOC + SHF_WRITE;
	shdr->sh_name = 
		add_str(einfop->shstrscn_info, ".prim_plist");
	return;
}

static void
make_primordial_page_scn(elf_info_t *einfop, char *buf, size_t size)
{
	Elf_Scn *scn;		/* section descriptor */
	Elf_Data *data;		/* section data */
	Elf32_Shdr *shdr;	/* section header */
	Elf *elf = einfop->elf;
	ulong_t count = size / PAGESIZE;

	scn = elf_newscn(elf);
	shdr = elf32_getshdr(scn);

	/* create object in the .count scn to hold the page count */
	add_cnt(einfop->cntscn_info, "prim_pagecnt", sizeof(int *), 
		count, einfop->symscn_info);

	/* create a symbol to point to the pages data */
	add_symbol(einfop->symscn_info, "prim_pages", 0, size, scn);

	data = elf_newdata(scn);
	if ((data->d_buf = (char *) malloc(size)) == NULL)
		error("make_primordial_page_scn: "
			"Memory allocation failure");

	memcpy(data->d_buf, buf, size);
	data->d_size = size;
	data->d_off = 0;
	data->d_align = 0x1000;
	shdr->sh_type = SHT_PROGBITS;
	shdr->sh_flags = SHF_ALLOC + SHF_WRITE;
	shdr->sh_name = 
		add_str(einfop->shstrscn_info, ".prim_pages");
	return;
}

static void
add_cnt(scninfo_t *si, char *cntname, size_t size, uint_t value,
	scninfo_t *sym_si)
{
	Elf_Data *data;

	data = elf_newdata(si->scn);
	if ((data->d_buf = (char *) malloc(size)) == NULL)
		error("add_cnt: " "Memory allocation failure");

	memcpy(data->d_buf, &value, size);
	data->d_size = size;
	data->d_off = 0;
	data->d_align = 4;

	/* create a symbol to point to the node count */
	add_symbol(sym_si, cntname, si->offset, sizeof(int *), si->scn);
	si->offset += data->d_size;
}
static uint_t
add_str(scninfo_t *si, char *string)
{
	uint_t ret;

	if (si->shdr->sh_type != SHT_STRTAB)
		error("add_str: attempt to add a string(%s) to a "
			"non-string section", string);

	if ((si->offset + strlen(string) + 1) > si->data->d_size)
		error("add_str: String table overflow");

	strcpy((char *)si->data->d_buf + si->offset, string);
	ret = si->offset;
	si->offset += (strlen(string) + 1);
	return ret;
}

static void
add_symbol(scninfo_t *sym_si, char *string, int value, size_t size, 
	Elf_Scn *targ_scn)
{
	Elf32_Sym sym;

	if (sym_si->shdr->sh_type != SHT_SYMTAB) 
		error("add_symbol: Attempt to add symbol to a "
			"non-symbol table section");

	if ((sym_si->offset + sizeof(Elf32_Sym)) > sym_si->data->d_size)
		error("add_symbol: Symbol table overflow");

	sym.st_name = add_str(sym_si->link, string);
	sym.st_value = value;
	sym.st_size = size;
	sym.st_info = ELF32_ST_INFO(STB_GLOBAL, STT_OBJECT);
	sym.st_other = 0;
	sym.st_shndx = elf_ndxscn(targ_scn);

	memcpy((char *)sym_si->data->d_buf + sym_si->offset, 
		&sym, sizeof(Elf32_Sym));
	sym_si->offset += sizeof(Elf32_Sym);
	return;
}
	
static void
set_tab_size(scninfo_t *si)
{
	si->data->d_size = si->offset;
}

static int
readfile(
char *filename,	/* filename to read */
char *dst,	/* location into which data is read */
int maxsize)	/* maximum size to read */
{
	int fd;
	int len;

	if ((fd = open(filename, O_RDONLY)) == -1)
		error("readfile: Cannot open file: %s", filename);

	if ((len = lseek(fd, 0, SEEK_END)) == -1) {
		close(fd);
		error("readfile: Seek failure: %s", filename);
	}

	if (len > maxsize) 
		error("readfile: File size: 0x%X, larger than space allocated: "
                      "0x%X, File: %s", len, maxsize, filename);

	lseek(fd, 0, SEEK_SET);

	if ((len = read(fd, dst, maxsize)) == -1) {
		close(fd);
		error("readfile: Read failure: %s", filename);
	}
	close(fd);
	return len;
}

static char *search_path = ".";

static void
set_search_path(char *pathstr)
{
	search_path = pathstr;
}

/* If name does not include a '/', try to build a path for it based
 * on the search path. If the file does not exist in any of the
 * directories in the search path, panic.
 */
char *
getfullname(char *name)
{
        char *fname;
        char *cp;

        if (name == NULL | *name == '\0')
                return NULL;

	/* See if the filename has a '/'. If so, don't attempt to
	 * augment it.
	 */
	cp = name;
	while (*cp) {
		if (*cp++ == '/')
			if (access(name, F_OK) == 0) {
				return name;
			} else
				error("getfullname: Cannot find file: %s\n", name);
	}
        /* try the different search directories */
        cp = search_path;
        while (*cp) {
		char *tp = cp;
                char savech;

                while (*tp && *tp != ':') /* get to colon */
                        tp++;
                savech = *tp;
                *tp = '\0';
                fname = (char *)malloc(strlen(name) + strlen(cp) + 2);
                strcpy(fname, cp);
                strcat(fname, "/");
                strcat(fname, name);
                *tp = savech;
                if (access(fname, F_OK) == 0) {
                        return fname;
		}
                free(fname);
                cp = tp + 1;
        }
	error("getfullname: Cannot find file: %s\n", name);
}

static void
error(char *fmt, ...)
{
	va_list	args;

	va_start(args, fmt);
	fprintf(stderr, "%s: ", cmdname);
	(void) vfprintf(stderr, fmt, args);
	va_end(args);
	fprintf(stderr, "\n");
	exit(1);
}

