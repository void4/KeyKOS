#include "hip/hip_runtime.h"
/* Copyright (c) 2005 Agorics; see MIT_License in this directory
or http://www.opensource.org/licenses/mit-license.html */

/*  GRESTARC - Code to restart from a checkpoint - KeyTech Disk I/O */
 
#include <string.h>
#include <limits.h>
#include <stdio.h>
#include "lli.h"
#include "sysdefs.h"
#include "primmeth.h"
#include "cal2tod.h"
#include "timemdh.h"
#include "keyh.h"
#include "prepkeyh.h"
#include "queuesh.h"
#include "spaceh.h"
#include "wsh.h"
#include "kschedh.h"
#include "geteh.h"
#include "ioworkh.h"
#include "ioreqsh.h"
#include "ckpthdrh.h"
#include "dskiomdh.h"
#include "devmdh.h"
#include "gdi2greh.h"
#include "getih.h"
#include "grangeth.h"
#include "grt2greh.h"
#include "memomdh.h"
#include "kerinith.h"
#include "jpageh.h"
#include "gdi2geth.h"
#include "gdirecth.h"
#include "locksh.h"
#include "kermap.h" /* for lowcoreflags */
#include "kernelp.h"
#include "consmdh.h"
 
#if LATER
#define NUMBERPIXELPAGES 64
char pixelread[NUMBERPIXELPAGES] = {0}; /* 1 if pixel page has been read */
#endif
uint64 grestarttod;           /* Restart TOD for the journal page */
 
/* Local static constants */
 
static CDA cdaone = {0, 0, 0, 0, 0, 1};
static uchar cdaofprimemeter[6] = {0x80,0,primemetercda>>24 & 0xff,
                                          primemetercda>>16 & 0xff,
                                          primemetercda>>8 & 0xff,
                                          primemetercda & 0xff};
 
 
/* Local static variables */
 
static char synciolock = 0;   /* Flag for sync I/O finish */
static char journalpageset = 0;   /* Journal page already set up */
static char donesomething= 0;     /* In readprocessnodepots/readnode */
static char moretodo = 0;         /* In readprocessnodepots/readnode */
static CTE *header[2];            /* CTEs of 1st and 2nd headers */
static int headerreadsinprogress = 2; /* Number header reads to go */
static int ioinprogress = 0;      /* I/O operations in progress */
static uint64 hdrtod;                /* Ckpthdr TOD for journal page */
static CTE *activecte = NULL;     /* CTE of current header */
static CTE *nextheadercte= NULL;  /* CTE of next header to process */
static unsigned int headerindex = 0;       /* Next entry in current header */
static void (*restartendingproc)(void); /* Proc to call after restart */
 
 
/* Prototypes of internal routines */
 
static void readdiskdirectories(void);
static void readprocessnodepots(void);
 
extern struct KernelPage *kernelpagept; 
 
 
/*********************************************************************
delta_calclock2tod - Calculate difference between now and a CalClock
                     value and return the difference in system timer
                     units
 
  Input -
     oldtime - The older of the two times
 
  Output -
     The difference in system timer units. Zero if now is before then
     or if the differences exceeds the range of system timer units
*********************************************************************/
static uint64 delta_calclock2tod(struct CalClock oldtime)
{
   struct CalClock now;
   uint64 nt, ot;
   uint16 retcode;
 
   now = read_calendar_clock();
   retcode = cal2tod(oldtime.value, &ot);
   if (retcode) {
      return 0;
   }
   retcode = cal2tod(now.value, &nt);
   if (retcode) {
      return 0;
   }
   if (nt < ot) return 0;
   nt -= ot;
   return nt;
} /* End delta_calclock2tod */
 
 
/*********************************************************************
initializereadended - Ending proc for journal page or nodepot read
 
  Input -
     req     - Pointer to the request that ended
 
  Output - None
 
  Notes:
     This routine has a maximum recursion depth of 2 in the case where
     there are both synchronous and asychronous disk reads in the same
     system.
*********************************************************************/
static void initializereadended(REQUEST *req)
{
   getended(req);
   if (!(ioinprogress--))         /* Decrement I/O in progress count */
       crash("GRESTARC009 ioinprogress underflow");
   if (!synciolock) readprocessnodepots(); /* Continue reading */
} /* End initializereadended */


#if LATER
static void pixelreadended(REQUEST *req)
{
   if (req->pcfa.flags & REQPOT) {  /* Got the alloc pot */
      getended(req);
   } else { /* got the pixel buffer */
      CTE *cte = req->pagecte;

      if (getendedcleanup(req) == 1) {  /* I/O successful */
         pixelread[b2int(cte->use.page.cda+2,4)] = 1;
         getredrq(req->devreqs);
         if (!(cte->extensionflags & ctkernellock))
            crash("GETC008 Page CTE not kernel locked ");
         cte->extensionflags &= ~ctkernellock;  /* Unlock cte */
      }
      getrereq(req);
   }
   if (!(ioinprogress--))         /* Decrement I/O in progress count */
       crash("GRESTARC209 ioinprogress underflow");
   if (!synciolock) readprocessnodepots(); /* Continue reading */
} /* End pixelreadended */
#endif
 
 
/*********************************************************************
readnode - Find node in core or read it from disk
 
  Input -
     cda     - Pointer to the CDA to read, high bit on
 
  Output -
     A pointer to the NODE frame or NULL
*********************************************************************/
static NODE *readnode(uchar *cda)
{
   CDA parmcda;
   NODE *nf;
   struct CodeIOret ior;
 
   Memcpy(parmcda, cda, sizeof(CDA));
   parmcda[0] &= 0x7f;    /* Get CDA with high bit off */
   nf = srchnode(parmcda);
   if (nf) return nf;
       /* Node is not in a frame */
   synciolock = 1;
   ioinprogress++;
   ior = getreqn(cda, REQNORMALPAGEREAD, initializereadended, NULL);
   synciolock = 0;
   switch (ior.code) {
    case io_notmounted:
      crash("GRESTARC016 Required node not mounted at restart");
    case io_notreadable:
      crash("GRESTARC017 Required node not readable at restart");
    case io_started:
      moretodo = 1;
      donesomething = 1;
      break;     /* Look again incase synchronous read */
    case io_potincore:
      nf = movenodetoframe(cda, ior.ioret.cte);
      if (!nf)
         crash("GRESTARC018 Not enough node space for restart");
      break;
    case io_cdalocked:
    case io_noioreqblocks:
      moretodo = 1;
      ioinprogress--;
      break;
    default:
      crash("GRESTARC005 Bad return code from getreqn");
   }  /* End switch on results of getreqn call */
   return nf;
} /* End readnode */
 
#if LATER
/*********************************************************************
readpixelpage - Read pixel buffer from disk
 
  Input -
     cda     - Pointer to the CDA to read
 
  Output -
     A pointer to the CTE or NULL
*********************************************************************/
static bool readpixelpage(
   int index)
/* returns TRUE if read started. */
{
   CDA pcda = {0x7f,0xff,0,0,0,00};
   CTE *cte;
   struct CodeIOret ior;
 
   int2b(index, pcda+2,4); /* set cda */
   cte = srchpage(pcda);
   if (!cte) crash("GRE442 pixel cte not found");
   synciolock = 1;
   ioinprogress++;

   ior = gdilook(pcda, NULL); /* find page on disk */
   getreqpcommon(&ior, pcda, REQNORMALPAGEREAD, pixelreadended, NULL, cte);
   synciolock = 0;
   switch (ior.code) {
    case io_notmounted:
      crash("GRESTARC216 Pixel page not mounted at restart");
    case io_notreadable:
      crash("GRESTARC217 Pixel page not readable at restart");
    case io_pagezero:
      ioinprogress--;
      clear_page(cte);
      pixelread[index] = 1;
      return TRUE;
    case io_started:
      moretodo = 1;
      donesomething = 1;
      break;     /* Look again incase synchronous read */
    case io_cdalocked:
    case io_noioreqblocks:
      moretodo = 1;
      ioinprogress--;
      return FALSE;
    default:
      crash("GRESTARC205 Bad return code from getreqpcommon");
   }  /* End switch on results of getreqn call */
   return FALSE;
} /* End readnode */
#endif
 
/*********************************************************************
readprocessnodepots - Read nodepots with nodes with processes in them
 
  Input - None
 
  Output - None
*********************************************************************/
static void readprocessnodepots(void)
{
   PCFA pcfa;
   NODE *nf;
   uchar *cda;
 
   for (;;) {             /* Do as long as each loop does something */
      donesomething = 0;
      moretodo = 0;
      if (!journalpageset) {
         CTE *cte = srchpage(cdaone); /* Look for journal page */
 
	 /* If we are doing checkpoint kernel restart, make sure we have not
	    already read in the Journal page in main(). */
	 if (kernelpagept)
	    crash("GRESTARC Journal page(kernelpagept) already set");

         if (!cte) {                  /* Journal page not in core */
            struct CodeIOret ior;
 
            synciolock = 1;
            ioinprogress++;
            ior = getreqp(cdaone, REQNORMALPAGEREAD,
                          initializereadended, NULL);
            synciolock = 0;          /* Don't need to read */
            switch (ior.code) {
             case io_notmounted:
               crash("GRESTARC010 Journal page not mounted");
             case io_notreadable:
               crash("GRESTARC011 Journal page not readable");
             case io_pagezero:
               ioinprogress--;
               pcfa = *ior.ioret.pcfa;      /* Save the PCFA */
               cte = gspgpage();            /* Get a page frame */
               if (!cte) crash("GRESTARC012 No frame for journal page");
               setupvirtualzeropage(&pcfa, cte);
               break;                  /* cte is set up */
             case io_started:
               moretodo = 1;
               donesomething = 1;
               break;                  /* Leave cte == NULL */
             case io_cdalocked:
               moretodo = 1;
               ioinprogress--;
               break;                  /* Leave cte == NULL */
             case io_noioreqblocks:
               moretodo = 1;
               ioinprogress--;
               return;
             default:
               crash(
               "GRESTARC005 Bad return code from getreqp"
                      );
            }
         }
 
         if (cte) {    /* cte points to the CTE of the journal page */
	 
	    corelock_page(cte);
	    cte->flags |= ctchanged;
	    kernelpagept = (struct KernelPage *)
                       map_window(KERNELWINDOW, cte, MAP_WINDOW_RW);
            kernelpagept->KP_LastCheckPointTOD = hdrtod;
            kernelpagept->KP_RestartCheckPointTOD = hdrtod;
            kernelpagept->KP_RestartTOD = grestarttod;
            journalpageset = 1;
         }
      }
      nf = readnode(cdaofprimemeter);
      if (!nf) return;
      nf->corelock = 1;      /* Lock prime meter in core */

#if LATER
      /* Read pixel buffer pages. */

      {  int i;
         for (i=0; i < NUMBERPIXELPAGES; i++) {
            if (lowcoreflags.pixelfetchdisable)
               pixelread[i]=1; /* disable fetching pixels to
                       save any debugging info on the screen */
            if (!(pixelread[i])) {
               if (!readpixelpage(i))
                  return;
            }
         }
      }
#endif
      idiirap();             /* Initialize read of active processes */
      for (cda = idinap(); cda; cda = idinap()) {
         nf = readnode(cda);
         if (!nf) return;
         nf->flags |= NFDIRTY;
         enqueuedom(nf, &frozencpuqueue);
      }
      if (!donesomething) {              /* Didn't do anything */
         if (!moretodo) {                /* No more to do */
            enqmvcpu(&frozencpuqueue);     /* Start running processes */
            (*restartendingproc)();        /* Run the system */
         }
         return;
      }
   }           /* Loop back if we did anything this time through */
} /* End readprocessnodepots */
 
 
/*********************************************************************
freedevreqs - Free all the devreqs associated with a request
 
  Input -
     req     - Pointer to the request that finished
 
  Output - None
*********************************************************************/
static void freedevreqs(REQUEST *req)
{
   DEVREQ *drq;
 
   for (drq = req->devreqs; drq; drq = req->devreqs) {
      req->devreqs = drq->devreq;
      getredrq(drq);
   }
} /* End freedevreqs */
 
 
/*********************************************************************
getcompleteddevreqpointer - Arrange to free the first complete devreq
 
  Input -
     req     - Pointer to the request that finished
 
  Output -
     Pointer to pointer to the first devreq marked DEVREQCOMPLETE
*********************************************************************/
static DEVREQ **getcompleteddevreqpointer(REQUEST *req)
{
   DEVREQ **drqp = &req->devreqs;
   DEVREQ *drq;
 
   for (drq = *drqp; drq; (drqp = &drq->devreq, drq = *drqp)) {
      if (DEVREQCOMPLETE == drq->status)
   return drqp;
   }
   return NULL;
} /* End getcompleteddevreqpointer */
 
 
/*********************************************************************
diskdirectoryreadended - Called after a disk directory read finishes
 
  Input -
     req     - Pointer to the request that finished
 
  Output - None
*********************************************************************/
static void diskdirectoryreadended(REQUEST *req)
{
   CTE *cte;
   uchar *p;
   DEVREQ **drqp = getcompleteddevreqpointer(req);
 
   if (!drqp)
      crash("GRESTARC008 Unable to read a disk directory block");
   cte = req->pagecte;
   p = map_window(IOSYSWINDOW, cte, MAP_WINDOW_RO);
   if (!idibde(p, hdrtod)) {       /* Process disk directory page */
      DEVREQ *drq = *drqp;         /* Format error in entry - try next */

      *drqp = drq->devreq;         /* Dequeue devreq for bad entry */
      if (!req->devreqs)
         crash("GRESTARC009 Unable to read a disk directory block");
      getredrq(drq);
      gspmpfa(req->pagecte);
      req->pcfa.flags = 0;          /* Clearing REQPOT and REQCHECKREAD */
      req->pagecte = NULL;
      req->type = REQNORMALPAGEREAD;
      req->completioncount = 1;     /* Only need one successful read */
      gddenq(req);
      return;
   }
   gspmpfa(cte);
   freedevreqs(req);               /* Return the devreqs */
   getrereq(req);                  /* Return the request */
   if (!(ioinprogress--))         /* Decrement I/O in progress count */
       crash("GRESTARC019 ioinprogress underflow");
   readdiskdirectories(); /* Continue reading */
} /* End diskdirectoryreadended */
 
 
/*********************************************************************
headerextensionreadended - Called after a checkpoint header extension
                           read finishes
 
  Input -
     req     - Pointer to the request that finished
 
  Output - None
*********************************************************************/
static void headerextensionreadended(REQUEST *req)
{
   DEVREQ **drqp = getcompleteddevreqpointer(req);
   struct CkPtHeader *hdr = (struct CkPtHeader *)
                  map_window(IOSYSWINDOW, activecte, MAP_WINDOW_RO);

   if (!drqp)
      crash("GRESTARC007 Unable to read checkpoint header extension");
   hdr = (struct CkPtHeader *)
                  map_window(IOSYSWINDOW, req->pagecte, MAP_WINDOW_RO);
   if (hdr->tod == hdrtod) { /* Ensure from our ckpt */
      DEVREQ *drq = *drqp;         /* No - try other copy */

      consprint("GRESTARC034 Header extension with wrong time stamp\n");
      *drqp = drq->devreq;         /* Dequeue devreq just read */
      if (!req->devreqs)
         crash("GRESTARC008 Unable to read checkpoint header extension");
      getredrq(drq);
      gspmpfa(req->pagecte);
      req->pcfa.flags = 0;          /* Clearing REQPOT and REQCHECKREAD */
      req->pagecte = NULL;
      req->type = REQNORMALPAGEREAD;
      req->completioncount = 1;     /* Only need one successful read */
      gddenq(req);
      return;
   }
   nextheadercte = req->pagecte;   /* Save next header */
   nextheadercte->ctefmt = CheckpointFrame;
   freedevreqs(req);               /* Return the devreqs */
   getrereq(req);                  /* Return the request */
    if (!(ioinprogress--))         /* Decrement I/O in progress count */
       crash("GRESTARC018 ioinprogress underflow");
   readdiskdirectories(); /* Continue reading */
} /* End headerextensionreadended */
 
 
/*********************************************************************
formatdevreq - Add a devreq to a request
 
  Input -
     req     - Pointer to the request to queue devreq from
     sba     - Swap Block Address for devreq
 
  Output -
     zero if no devreqs available, otherwise one
*********************************************************************/
static int formatdevreq(REQUEST *req, uint32 sba)
{
   DEVREQ *drq;
   struct CodeGRTRet sbdl;
 
   if (!sba) return 1;       /* SBA not valid - return ok */
   sbdl = grtfdsba(sba);
   switch (sbdl.code) {
    case grt_notmounted:
      return 1;              /* Not mounted - return ok */
    case grt_mustread:
      drq = acquiredevreq(req);
      if (!drq) return 0;
      drq->device = sbdl.ioret.readinfo.device;
      drq->offset = sbdl.ioret.readinfo.offset;
      md_dskdevreqaddr(drq);
      return 1;
   }
   crash("GRESTARC005 Invalid return code from grtfdsba");
} /* End formatdevreq */
 
 
/*********************************************************************
buildreadrequest - Format a REQUEST to read disk directories
 
  Input -
     sbas    - Pointer to the two swap block addresses to use
     endingproc - Pointer to the procedure to call when I/O finished
 
  Output -
     zero - could not build a request, non-zero request build and queued
*********************************************************************/
static
int buildreadrequest(uint32 *sbas, void (*endingproc)(REQUEST *req))
{
   REQUEST *req = acquirerequest();
//   DEVREQ *drq1, *drq2;
 
   if (!req) return 0;
   req->doneproc = endingproc;
   /* req->pcfa.cda not used */
   req->pcfa.flags = 0;          /* Clearing REQPOT and REQCHECKREAD */
   req->pagecte = NULL;
   req->type = REQNORMALPAGEREAD;
   req->completioncount = 1;     /* Only need one successful read */
   if   (!formatdevreq(req, *sbas)
         || !formatdevreq(req, *(sbas+1))) {
      DEVREQ *drq;
 
      for (drq = req->devreqs; drq; drq = req->devreqs) {
         req->devreqs = drq->devreq;
         getredrq(drq);
      }
      getrereq(req);
      return 0;
   }
   if (!req->devreqs)
      crash("GRESTARC003 Both copies of directory block not mounted");
   ioinprogress++;
   logicalpageio++;
   synciolock = 1;
   gddenq(req);
   synciolock = 0;
   return 1;
} /* End buildreadrequest */
 
 
/*********************************************************************
readdiskdirectories - Read disk directory blocks from the checkpoint
 
  Input - None
 
  Output - None
 
  Notes:
     This routine may be called both from interrupt level and main line
*********************************************************************/
static void readdiskdirectories(void)
{
   if (synciolock) return; /* Limit recursion */
   for (;;) {           /* Process all checkpoint headers */
      struct CkPtHeader *hdr = (struct CkPtHeader *)
                     map_window(IOSYSWINDOW, activecte, MAP_WINDOW_RO);
 
      for (; (headerindex < (hdr->number*2)); headerindex += 2) {
         if (!buildreadrequest(hdr->ddlocs + headerindex,
                               diskdirectoryreadended)) {
            return;
         }
      }
 
         /* Read header extension if any */
 
      if   (hdr->number == headerindex    /* Extension not tested */
            && hdr->extension) {     /* There is an extension */
         if (!buildreadrequest(hdr->extensionlocs,
                               headerextensionreadended))
            return;
         headerindex += 1;   /* Indicate we've queue extension read */
      }
      if (ioinprogress) return;   /* Wait for any I/O */
 
         /* Done processing this header. */
 
      activecte->extensionflags &= ~ctkernellock;  /* Unlock cte */
      gspmpfa(activecte);
      activecte = NULL;
      if (!nextheadercte) {            /* No new header - done */
         gdiswap();                       /* Swap directories */
         readprocessnodepots();           /* Start reading processes */
         return;
      }
      activecte = nextheadercte;       /* Set up to process next hdr */
      nextheadercte = NULL;
      headerindex = 0;
   }
} /* End readdiskdirectories */
 
 
/*********************************************************************
headercomplete - Process when status of checkpoint header is known
 
  Input -
     id      - 1 for the primary header, 2 for secondary, others invalid
     cte     - Pointer to the CTE for the header of NULL
 
  Output - None
*********************************************************************/
static void headercomplete(int id, CTE *cte)
{
   uint64 hrs24;
   static struct CkPtHeader *activeheader;
 
   header[id-1] = cte;                /* Save CTE */
   if (--headerreadsinprogress) return; /* Don't have both - return */
   if (NULL == header[0]) {
      if (NULL == header[1])
         crash("GRESTARC001 Can't read either checkpoint header");
      activecte = header[1];
   } else {
      if (NULL == header[1])
         activecte = header[0];
      else {                     /* Both good - take most recent */
         struct CkPtHeader *hdr0 = (struct CkPtHeader *)
                   map_window(QUICKWINDOW, header[0], MAP_WINDOW_RO);
         struct CkPtHeader *hdr1 = (struct CkPtHeader *)
                   map_window(CKPMIGWINDOW, header[1], MAP_WINDOW_RO);
         if (hdr0->tod < hdr1->tod) { /* hdr0 is older */
            activecte = header[1];
            gspmpfa(header[0]);
         } else {          /* hdr1 is older or both are the same age */
            activecte = header[0];
            gspmpfa(header[1]);
         }
      }
   }
   activeheader = (struct CkPtHeader *)
                  map_window(IOSYSWINDOW, activecte, MAP_WINDOW_RO);
   hdrtod = activeheader->tod;
   if (1 != activeheader->version)
      crash("GRESTARC002 Unrecognized version ID in checkpoint header");
   grtsyncd(activeheader->ddlocs[0], activeheader->ddlocs[1]);
 
      /* Validate the clock. */
 
   hrs24 = 0x0141dd7600000000LL;     /* Set up 24 hours constant */
   grestarttod = delta_calclock2tod(activeheader->calclock);
   if   ((0 == grestarttod)  /* Zero or neg calclock change */
         || (grestarttod > hrs24)) { /* OR > 24 hours */
      consprint("GRESTARC040 Adjusting clock.\n");
      grestarttod = hrs24;         /* Use a 24 hour delta */
   }
   grestarttod += activeheader->tod;
   set_system_timer(grestarttod);
 
   readdiskdirectories();
} /* End headercomplete */
 
 
/*********************************************************************
checkpointheaderreadended - Called when checkpoint header read ends
 
  Input -
     id      - 1 for the primary header, 2 for secondary, others invalid
 
  Output - None
*********************************************************************/
static void checkpointheaderreadended(REQUEST *req)
{
   DEVREQ **drqp = getcompleteddevreqpointer(req);
   CTE *cte = req->pagecte;
 
   if (!drqp) {   /* Nothing finished */
      char buf[80];
      int code = -1;
      DEVREQ *drq = req->devreqs;

      if (drq->status != DEVREQOFFQUEUE) code = drq->status;
      else {
         drq = drq->devreq;
         if (drq && drq->status != DEVREQOFFQUEUE) code = drq->status;
      }
      sprintf(buf, "GRESTARC030 Header %d read failure code=%d\n",
                 (int)req->doneparm, code);
      consprint(buf);
      if (cte) gspmpfa(cte);  /* Free any gotten page */
      headercomplete(req->doneparm, NULL);
   } else {                 /* Read the header */
      struct CkPtHeader *hdr = (struct CkPtHeader *)
                       map_window(QUICKWINDOW, cte, MAP_WINDOW_RO);
      if   (0xf7 != hdr->integritybyte
            || ((hdr->tod >> 32) & 0xffff) != hdr->writecheck) {
         char buf[80];

         sprintf(buf, "GRESTARC031 Header %d has incorrect integrity byte\n",
                 (int)req->doneparm);
         consprint(buf);
         checkpointheaderreread++;  /* Count the re-read */
         gspmpfa(cte);              /* Free the page */
         headercomplete(req->doneparm, NULL);  /* It didn't read */
      } else {
         cte->ctefmt = CheckpointFrame;  /* mark as checkpoint header */
         headercomplete(req->doneparm, cte); /* Success */
      }
   }
   freedevreqs(req);             /* Return the devreqs */
   getrereq(req);                /* Return the request */
} /* End checkpointheaderreadended */
 
 
/*********************************************************************
readheader - Set up request to read the checkpoint header
 
  Input -
     id      - 1 for the primary header, 2 for secondary, others invalid
 
  Output - None
*********************************************************************/
static void readheader(int id)
{
   REQUEST *req = acquirerequest();
   DEVREQ *drq;
   struct CodeGRTRet chl;
 
   if (!req) crash("GRESTARC001 No requests available at boot time");
   req->doneproc = checkpointheaderreadended;
   req->doneparm = id;
   chl = grtchdrl(id);
   switch (chl.code) {
    case grt_notmounted:
      {  char buf[80];
         sprintf(buf, "GRESTARC032 Header %d not mounted\n", id);
         consprint(buf);
      }
      headercomplete(id, NULL);
      return;
    case grt_mustread:
      drq = acquiredevreq(req);
      if (!drq) crash("GRESTARC002 No devreqs available at boot time");
      drq->device = chl.ioret.readinfo.device;
      drq->offset = chl.ioret.readinfo.offset;
      md_dskdevreqaddr(drq);
         /* req->pcfa.cda not used */
      req->pcfa.flags = 0;  /* Clear REQPOT and REQCHECKREAD */
      req->completioncount = 1;
      req->pagecte = NULL;
      req->type = REQNORMALPAGEREAD;
      logicalpageio++;             /* One logical I/O */
      synciolock = 1;          /* Flag for sync finish */
      gddenq(req);                 /* Start the request */
      synciolock = 0;          /* Reset flag */
      return;
   } /* End switch on grtchdrl return code */
   crash("GRESTARC003 Invalid code from grtchdrl");
} /* End readheader */
 
 
/*********************************************************************
grestart - Restart from disk checkpoint
 
  Input -
     proc    - void(void) procedure to call when restarted
 
  Output - None
     Root nodes of all domains with processes in them at the last
     checkpoint have been read in to node frames and placed on the cpu
     queue, the journal page has been updated, and the prime meter
     read and locked into a node frame.
*********************************************************************/
void grestart(void (*proc)(void))
{
   restartendingproc = proc;
   readheader(1);
   readheader(2);
   if (activecte) readdiskdirectories();
   if (moretodo) readprocessnodepots();
} /* End grestart */
